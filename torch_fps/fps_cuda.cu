#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>

#include <c10/util/Exception.h>

#include <limits>
#include <type_traits>

namespace torch_fps {
namespace {

template <typename scalar_t, typename acc_t, int BLOCK_SIZE, int MAX_D>
__global__ void fps_kernel_cuda(
    const scalar_t* __restrict__ points,
    const bool* __restrict__ mask,
    const int64_t* __restrict__ start_idx,
    int64_t B,
    int64_t N,
    int64_t D,
    int64_t K,
    int64_t* __restrict__ idx,
    acc_t* __restrict__ min_dists) {
    const int b = blockIdx.x;
    if (b >= B) {
        return;
    }

    const scalar_t* batch_points = points + static_cast<int64_t>(b) * N * D;
    const bool* batch_mask = mask + static_cast<int64_t>(b) * N;
    const int64_t start = start_idx[b];
    int64_t* batch_idx = idx + static_cast<int64_t>(b) * K;
    acc_t* batch_min_dists = min_dists + static_cast<int64_t>(b) * N;

    __shared__ int64_t shared_last;
    __shared__ int64_t shared_counts[BLOCK_SIZE];
    __shared__ acc_t shared_vals[BLOCK_SIZE];
    __shared__ int64_t shared_idx[BLOCK_SIZE];

    if (threadIdx.x == 0) {
        shared_last = (start >= 0 && start < N) ? start : 0;
    }
    __syncthreads();

    const acc_t inf = std::numeric_limits<acc_t>::infinity();

    int64_t local_count = 0;
    for (int64_t n = threadIdx.x; n < N; n += BLOCK_SIZE) {
        const bool valid = batch_mask[n];
        if (valid) {
            batch_min_dists[n] = inf;
            ++local_count;
        } else {
            batch_min_dists[n] = acc_t(0);
        }
    }

    shared_counts[threadIdx.x] = local_count;
    __syncthreads();

    for (int offset = BLOCK_SIZE / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            shared_counts[threadIdx.x] += shared_counts[threadIdx.x + offset];
        }
        __syncthreads();
    }

    const int64_t valid_count = shared_counts[0];
    const int64_t effective_k = valid_count < K ? valid_count : K;
    int64_t last = shared_last;

    if (K == 0) {
        return;
    }

    if (effective_k == 0) {
        if (threadIdx.x == 0) {
            for (int64_t i = 0; i < K; ++i) {
                batch_idx[i] = last;
            }
        }
        return;
    }

    for (int64_t i = 0; i < K; ++i) {
        if (threadIdx.x == 0) {
            batch_idx[i] = last;
        }
        __syncthreads();

        if (i + 1 >= effective_k) {
            continue;
        }

        acc_t centroid_vals[MAX_D];
        for (int64_t d = 0; d < D; ++d) {
            centroid_vals[d] = static_cast<acc_t>(batch_points[last * D + d]);
        }

        acc_t best_val = -std::numeric_limits<acc_t>::infinity();
        int64_t best_idx = last;

        for (int64_t n = threadIdx.x; n < N; n += BLOCK_SIZE) {
            if (!batch_mask[n]) {
                continue;
            }

            const scalar_t* point = batch_points + n * D;
            acc_t dist = acc_t(0);
            for (int64_t d = 0; d < D; ++d) {
                const acc_t diff =
                    static_cast<acc_t>(point[d]) - centroid_vals[d];
                dist += diff * diff;
            }

            acc_t current = batch_min_dists[n];
            if (dist < current) {
                current = dist;
            }
            batch_min_dists[n] = current;

            if (current > best_val ||
                (current == best_val && n < best_idx)) {
                best_val = current;
                best_idx = n;
            }
        }

        shared_vals[threadIdx.x] = best_val;
        shared_idx[threadIdx.x] = best_idx;
        __syncthreads();

        for (int offset = BLOCK_SIZE / 2; offset > 0; offset >>= 1) {
            if (threadIdx.x < offset) {
                const acc_t other_val = shared_vals[threadIdx.x + offset];
                const int64_t other_idx = shared_idx[threadIdx.x + offset];
                const acc_t current_val = shared_vals[threadIdx.x];
                const int64_t current_idx = shared_idx[threadIdx.x];

                if (other_val > current_val ||
                    (other_val == current_val && other_idx < current_idx)) {
                    shared_vals[threadIdx.x] = other_val;
                    shared_idx[threadIdx.x] = other_idx;
                }
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            last = shared_idx[0];
            shared_last = last;
        }
        __syncthreads();
    }
}

}  // namespace

at::Tensor fps_forward_cuda(
    const at::Tensor& points,
    const at::Tensor& mask,
    const at::Tensor& start_idx,
    int64_t K) {
    TORCH_CHECK(points.is_cuda(), "points tensor must be on CUDA");
    TORCH_CHECK(mask.is_cuda(), "mask tensor must be on CUDA");
    TORCH_CHECK(start_idx.is_cuda(), "start_idx tensor must be on CUDA");

    TORCH_CHECK(points.dim() == 3, "points tensor must have shape [B, N, D]");
    TORCH_CHECK(mask.sizes() == at::IntArrayRef({points.size(0), points.size(1)}),
                "mask tensor must have shape [B, N]");
    TORCH_CHECK(start_idx.numel() == points.size(0),
                "start_idx tensor must have shape [B]");

    TORCH_CHECK(points.scalar_type() == at::kFloat || points.scalar_type() == at::kDouble,
                "points tensor must be float32 or float64");
    TORCH_CHECK(mask.scalar_type() == at::kBool,
                "mask tensor must be boolean");

    TORCH_CHECK(K >= 0, "K must be non-negative");

    auto points_contig = points.contiguous();
    auto mask_contig = mask.contiguous();
    auto start_contig = start_idx.contiguous();

    const auto B = points_contig.size(0);
    const auto N = points_contig.size(1);
    const auto D = points_contig.size(2);

    TORCH_CHECK(D <= 16,
                "torch-fps CUDA kernel supports up to 16 feature dimensions");

    auto idx = at::empty({B, K},
                         at::TensorOptions()
                             .dtype(at::kLong)
                             .device(points_contig.device()));

    constexpr int BLOCK_SIZE = 256;
    constexpr int MAX_D = 16;
    const dim3 blocks(static_cast<unsigned int>(B));

    AT_DISPATCH_FLOATING_TYPES(points_contig.scalar_type(), "fps_forward_cuda", [&] {
        using acc_t = at::acc_type<scalar_t, true>;
        const at::ScalarType acc_scalar_type =
            std::is_same<acc_t, double>::value ? at::kDouble : at::kFloat;
        auto min_dists =
            at::empty({B, N}, points_contig.options().dtype(acc_scalar_type));
        fps_kernel_cuda<scalar_t, acc_t, BLOCK_SIZE, MAX_D>
            <<<blocks, BLOCK_SIZE, 0, at::cuda::getCurrentCUDAStream()>>>(
                points_contig.data_ptr<scalar_t>(),
                mask_contig.data_ptr<bool>(),
                start_contig.data_ptr<int64_t>(),
                B,
                N,
                D,
                K,
                idx.data_ptr<int64_t>(),
                min_dists.data_ptr<acc_t>());
    });

    AT_CUDA_CHECK(hipGetLastError());

    return idx;
}

}  // namespace torch_fps
